#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <algorithm>
#include <functional>
#include <vector>   
#include <cmath>     
#include "a3.hpp"

#define M_PI 3.14159265358979323846

__global__ void kde_kernel(int n, float h, const float* x, float* y) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (idx < n) {
        double sum = 0.0;
        float xi = x[idx];
        const double inv_h = 1.0 / h;
        const double norm_factor = 1.0 / (sqrtf(2.0 * M_PI) * h * n);
        
        for (int j = 0; j < n; j++) {
            double diff = (xi - x[j]) * inv_h;
            double kernel = expf(-0.5 * diff * diff);
            sum += kernel;
        }

        __syncthreads();
        
        y[idx] = sum * norm_factor;
    }
}


void gaussian_kde(int n, float h, const std::vector<float>& x, std::vector<float>& y) {
    float *d_x, *d_y;
    hipMalloc(&d_x, n * sizeof(float));
    hipMalloc(&d_y, n * sizeof(float));
    
    hipMemcpy(d_x, x.data(), n * sizeof(float), hipMemcpyHostToDevice);
    
    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    
    kde_kernel<<<blocksPerGrid, threadsPerBlock>>>(n, h, d_x, d_y);
    
    hipMemcpy(y.data(), d_y, n * sizeof(float), hipMemcpyDeviceToHost);
    
    hipFree(d_x);
    hipFree(d_y);
}
